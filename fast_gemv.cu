#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <driver_functions.h>
#include <stdio.h>

#include "fast_gemv.cuh"

#define WARP_SIZE 32

struct __align__(8) half4 { half x, y, z, w; };

// one block per 4 rows (gridDim.x = 1, gridDim.y = 128)
// thread_per_block = blockDim.x = WARP_SIZE
__global__ void gemv_fp16_512(half* mat, half* vec, half* res, unsigned int n,
                              unsigned int num_per_thread) {
  half sum = 0;
  // each thread load num_per_thread elements from global
  unsigned int tid = threadIdx.x;
  unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int start_idx = threadIdx.x;
  half4* mat4 = reinterpret_cast<half4*>(mat);
  half4* vec4 = reinterpret_cast<half4*>(vec);

#pragma unroll
  for (int iter = 0; iter < num_per_thread >> 2; iter++) {
    unsigned int j = start_idx + iter * blockDim.x;
    if (j < n >> 2) {
      half4 vec_val = vec4[j];
      half4 mat_val = mat4[row * (n >> 2) + j];
      sum += vec_val.x * mat_val.x;
      sum += vec_val.y * mat_val.y;
      sum += vec_val.z * mat_val.z;
      sum += vec_val.w * mat_val.w;
    }
  }

  sum = warpReduceSum(sum, blockDim.x);

  if (tid == 0) {
    res[row] = sum;
  }
}

// thread_per_block = blockDim.x = WARP_SIZE
__global__ void gemv_fp16_16384(half* mat, half* vec, half* mid_res,
                                unsigned int n, unsigned int num_per_thread) {
  half sum = 0;
  // each thread load num_per_thread elements from global
  unsigned int tid = threadIdx.x;
  unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int start_idx =
      blockIdx.x * (blockDim.x * num_per_thread) / 4 + threadIdx.x;
  half4* mat4 = reinterpret_cast<half4*>(mat);
  half4* vec4 = reinterpret_cast<half4*>(vec);

#pragma unroll
  for (int iter = 0; iter < num_per_thread / 4; iter++) {
    unsigned int j = start_idx + iter * blockDim.x;
    if (j < n / 4) {
      half4 vec_val = vec4[j];
      half4 mat_val = mat4[row * (n / 4) + j];
      sum += vec_val.x * mat_val.x;
      sum += vec_val.y * mat_val.y;
      sum += vec_val.z * mat_val.z;
      sum += vec_val.w * mat_val.w;
    }
  }

  sum = warpReduceSum(sum, blockDim.x);

  if (tid == 0) {
    mid_res[row * gridDim.x + blockIdx.x] = sum;
  }
}

// 32 blocks per 4 rows
// thread_per_block * num_per_thread = num_per_block = n / blockDim.x
__global__ void gemv_fp16(half* mat, half* vec, half* mid_res, unsigned int n,
                          unsigned int thread_per_block,
                          unsigned int num_per_thread) {
  half sum = 0;
  // each thread load num_per_thread elements from global
  unsigned int tid = threadIdx.x;
  unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int start_idx =
      blockIdx.x * (thread_per_block * num_per_thread) / 4 + threadIdx.x;
  half4* mat4 = reinterpret_cast<half4*>(mat);
  half4* vec4 = reinterpret_cast<half4*>(vec);

  // // Allocate shared memory for vec4
  // __shared__ half4 shared_vec4[128];

  // // Load vec4 into shared memory
  // if (threadIdx.y == 0) {
  //   for (int iter = 0; iter < num_per_thread / 4; iter++) {
  //     unsigned int j = start_idx + iter * thread_per_block;
  //     if (j < n / 4) {
  //       shared_vec4[threadIdx.x + iter * thread_per_block] = vec4[j];
  //     }
  //   }
  // }
  // __syncthreads();

#pragma unroll
  for (int iter = 0; iter < num_per_thread / 4; iter++) {
    unsigned int j = start_idx + iter * thread_per_block;
    if (j < n / 4) {
      // half4 vec_val = shared_vec4[threadIdx.x + iter * thread_per_block];
      half4 vec_val = vec4[j];
      half4 mat_val = mat4[row * (n / 4) + j];
      sum += vec_val.x * mat_val.x;
      sum += vec_val.y * mat_val.y;
      sum += vec_val.z * mat_val.z;
      sum += vec_val.w * mat_val.w;
    }
  }

  sum = warpReduceSum(sum, thread_per_block);

  if (thread_per_block <= WARP_SIZE) {
    if (tid == 0) {
      mid_res[row * gridDim.x + blockIdx.x] = sum;
    }
    return;
  }

  // Shared mem for partial sums (one per warp in the block)
  static __shared__ half warpLevelSums[WARP_SIZE];
  const int laneId = threadIdx.x % WARP_SIZE;
  const int warpId = threadIdx.x / WARP_SIZE;
  if (laneId == 0) warpLevelSums[warpId] = sum;
  __syncthreads();
  // read from shared memory only if that warp existed
  sum = (threadIdx.x < blockDim.x / WARP_SIZE) ? warpLevelSums[laneId]
                                               : (half)0.0;
  // Final reduce using first warp
  if (warpId == 0) sum = warpReduceSum(sum, thread_per_block / WARP_SIZE);
  if (tid == 0) {
    mid_res[row * gridDim.x + blockIdx.x] = sum;
  }
}

// block_num <= WARP_SIZE
__global__ void gemv_reduce_fp16(half* mid_res, half* res,
                                 unsigned int block_num) {
  half sum = 0;
  // each thread loads one element from global
  unsigned int tid = threadIdx.x;
  unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
  if (tid < block_num) {
    sum = mid_res[row * blockDim.x + tid];
  }
  sum = warpReduceSum(sum, block_num);
  if (tid == 0) {
    res[row] = sum;
  }
}

///////////////////////////// UTILITIES //////////////////////////////

__device__ __forceinline__ half warpReduceSum(half sum,
                                              unsigned int blockSize) {
  if (blockSize >= 32)
    sum += __shfl_down_sync(0xffffffff, sum, 16);  // 0-16, 1-17, 2-18, etc.
  if (blockSize >= 16)
    sum += __shfl_down_sync(0xffffffff, sum, 8);  // 0-8, 1-9, 2-10, etc.
  if (blockSize >= 8)
    sum += __shfl_down_sync(0xffffffff, sum, 4);  // 0-4, 1-5, 2-6, etc.
  if (blockSize >= 4)
    sum += __shfl_down_sync(0xffffffff, sum, 2);  // 0-2, 1-3, 4-6, 5-7, etc.
  if (blockSize >= 2)
    sum += __shfl_down_sync(0xffffffff, sum, 1);  // 0-1, 2-3, 4-5, etc.
  return sum;
}

__global__ void generate_random_numbers(half* numbers, int Np) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < Np) {
    hiprandState state;
    hiprand_init(clock64(), i, 0, &state);
    numbers[i] = __float2half(hiprand_uniform(&state));
  }
}

__global__ void generate_numbers(half* numbers, int Np) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < Np) {
    numbers[i] = __float2half(i / 100.0);
  }
}

__global__ void check_correctness(half* mat, half* vec, half* res, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    float result = 0;
    for (int j = 0; j < n; ++j) {
      result += __half2float(mat[idx * n + j]) * __half2float(vec[j]);
    }
    half half_result = __float2half(result);
    float diff = __half2float(res[idx]) - __half2float(half_result);
    float delta = 0.125 * n / 512;
    if (diff > delta || diff < -delta) {
      printf("!!![idx=%d] %f != %f, diff=%f\n", idx, __half2float(res[idx]),
             __half2float(result), diff);
    }
  }
}

// one thread for one dot product
__global__ void gemv_naive(half* mat, half* vec, half* res, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    float result = 0;
    for (int j = 0; j < n; ++j) {
      result += __half2float(mat[idx * n + j]) * __half2float(vec[j]);
    }
    res[idx] = __float2half(result);
  }
}