#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <driver_functions.h>
#include <hiprand/hiprand_kernel.h>

#include "fast_gemv.cuh"

__global__ void gemv_fp16_128(half* mat, half* vec, half* res, int n) {

}