#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <driver_functions.h>
#include <stdio.h>

#include "fast_gemv_quantized.cuh"

#define WARP_SIZE 32

struct half4 { half x, y, z, w; };
struct uint8_2 { uint8_t x, y; };
struct float4_2 { float4 x, y; };

// each thread computes 8 * 2(row) results
// gridDim.y = 128, blockDim.y = 2
__global__ void init_table_int8(half* vec, float* table, unsigned int n, float scale, int16_t zero_point) {
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int start_row = blockIdx.y * blockDim.y + threadIdx.y;
  float4* vec4 = reinterpret_cast<float4*>(vec);
  float4_2* table8 = reinterpret_cast<float4_2*>(table);
  if (idx >= n >> 3) {
    return;
  }
  float4 vec_val = vec4[idx];
  const half2* vec_h1 = (half2*)&vec_val.x;
  const half2* vec_h2 = (half2*)&vec_val.y;
  const half2* vec_h3 = (half2*)&vec_val.z;
  const half2* vec_h4 = (half2*)&vec_val.w;
  for (uint8_t i = start_row; i < start_row + blockDim.y; ++i) {
    float4_2 res;
    int8_t val = (int8_t)i;
    res.x.x = (scale * (val - zero_point)) * static_cast<float>(vec_h1->x);
    res.x.y = (scale * (val - zero_point)) * static_cast<float>(vec_h1->y);
    res.x.z = (scale * (val - zero_point)) * static_cast<float>(vec_h2->x);
    res.x.w = (scale * (val - zero_point)) * static_cast<float>(vec_h2->y);
    res.y.x = (scale * (val - zero_point)) * static_cast<float>(vec_h3->x);
    res.y.y = (scale * (val - zero_point)) * static_cast<float>(vec_h3->y);
    res.y.z = (scale * (val - zero_point)) * static_cast<float>(vec_h4->x);
    res.y.w = (scale * (val - zero_point)) * static_cast<float>(vec_h4->y);
    // write res to table
    table8[i * n / 8 + idx] = res;
  }
}

// num_per_thread >= 8
__global__ void gemv_quantized_int8_single_stage(int8_t* mat, half* res, float* table, unsigned int n,
                              unsigned int num_per_thread) {
  float sum = 0;
  // each thread load num_per_thread elements from global
  unsigned int tid = threadIdx.x;
  unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int start_idx = threadIdx.x;
  half4* mat4 = reinterpret_cast<half4*>(mat);

#pragma unroll
  for (int iter = 0; iter < num_per_thread >> 3; iter++) {
    unsigned int j = start_idx + iter * blockDim.x;
    if (j < n >> 3) {
      half4 mat_val = mat4[row * (n >> 3) + j];
      const uint8_2* mat_h1 = (uint8_2*)&mat_val.x;
      const uint8_2* mat_h2 = (uint8_2*)&mat_val.y;
      const uint8_2* mat_h3 = (uint8_2*)&mat_val.z;
      const uint8_2* mat_h4 = (uint8_2*)&mat_val.w;
      sum += table[mat_h1->x * n + j * 8];
      sum += table[mat_h1->y * n + j * 8 + 1];
      sum += table[mat_h2->x * n + j * 8 + 2];
      sum += table[mat_h2->y * n + j * 8 + 3];
      sum += table[mat_h3->x * n + j * 8 + 4];
      sum += table[mat_h3->y * n + j * 8 + 5];
      sum += table[mat_h4->x * n + j * 8 + 6];
      sum += table[mat_h4->y * n + j * 8 + 7];
    }
  }

  sum = warpReduceSum2(sum, blockDim.x);

  if (blockDim.x <= WARP_SIZE) {
    if (tid == 0) {
      res[row] = __float2half(sum);
    }
    return;
  }

  // Shared mem for partial sums (one per warp in the block)
  static __shared__ float warpLevelSums[32][WARP_SIZE];
  const int laneId = threadIdx.x % WARP_SIZE;
  const int warpId = threadIdx.x / WARP_SIZE;
  if (laneId == 0) warpLevelSums[threadIdx.y][warpId] = sum;
  __syncthreads();
  // read from shared memory only if that warp existed
  sum = (threadIdx.x < blockDim.x / WARP_SIZE) ? warpLevelSums[threadIdx.y][laneId] : 0.0;
  // Final reduce using first warp
  if (warpId == 0) sum = warpReduceSum2(sum, blockDim.x / WARP_SIZE);
  if (tid == 0) {
    res[row] = __float2half(sum);
  }
}

///////////////////////////// UTILITIES //////////////////////////////

__device__ __forceinline__ float warpReduceSum2(float sum,
                                               unsigned int blockSize) {
  if (blockSize >= 32)
    sum += __shfl_down_sync(0xffffffff, sum, 16);  // 0-16, 1-17, 2-18, etc.
  if (blockSize >= 16)
    sum += __shfl_down_sync(0xffffffff, sum, 8);  // 0-8, 1-9, 2-10, etc.
  if (blockSize >= 8)
    sum += __shfl_down_sync(0xffffffff, sum, 4);  // 0-4, 1-5, 2-6, etc.
  if (blockSize >= 4)
    sum += __shfl_down_sync(0xffffffff, sum, 2);  // 0-2, 1-3, 4-6, 5-7, etc.
  if (blockSize >= 2)
    sum += __shfl_down_sync(0xffffffff, sum, 1);  // 0-1, 2-3, 4-5, etc.
  return sum;
}

__global__ void generate_random_int8_numbers(int8_t* numbers, int Np) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < Np) {
    hiprandState state;
    hiprand_init(clock64(), i, 0, &state);
    numbers[i] = static_cast<int8_t>(hiprand(&state) % 128); // Random int8 number [-128, 127]
  }
}

__global__ void check_quantized_correctness(int8_t* mat, half* vec, half* res, float scale, int16_t zero_point, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    float result = 0;
    for (int j = 0; j < n; ++j) {
      float dequantized_val = (mat[idx * n + j] - zero_point) * scale;
      result += dequantized_val * __half2float(vec[j]);
    }
    half half_result = __float2half(result);
    float diff = __half2float(res[idx]) - __half2float(half_result);
    float delta = 0.125 * n / 512;
    if (diff > delta || diff < -delta) {
      printf("!!![idx=%d] %f != %f, diff=%f\n", idx, __half2float(res[idx]),
             __half2float(result), diff);
    }
  }
}
