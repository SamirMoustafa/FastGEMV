#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <driver_functions.h>
#include <math.h>
#include <stdio.h>

#include <cassert>
#include <chrono>

#include "utility.cuh"
#include "fast_gemv.cuh"
#include "simple_tensor.h"

///////////////////////////// SOLVER //////////////////////////////

SimpleTensor<half> solve_gemv_int4_quantized_with_params(const SimpleTensor<uint4_2>& mat, 
                                                    const SimpleTensor<half>& vec, 
                                                    unsigned int block_dim_x,
                                                    unsigned int block_dim_y, 
                                                    float scale_f, float zero_point_f) {
  half scale = __float2half(scale_f);
  half zero_point = __float2half(zero_point_f);
  assert(mat.width_ * 2 == vec.height_);
  assert(block_dim_y <= SHARED_MEM_MAX_ROWS);
  assert(block_dim_x * block_dim_y <= MAX_THREADS_PER_BLOCK);
  unsigned int num_per_thread = vec.height_ / block_dim_x;
  assert(num_per_thread >= 16);
  SimpleTensor<half> result(vec.height_, 1);
  dim3 grid_dim(1, mat.height_ / block_dim_y);
  dim3 block_dim(block_dim_x, block_dim_y);
  gemv_quantized_int4<<<grid_dim, block_dim>>>(mat.data_, vec.data_, result.data_, 
                                                            vec.height_, scale, zero_point, num_per_thread);
  checkCudaErrors(hipPeekAtLastError());
  return result;
}

SimpleTensor<half> solve_gemv_int8_quantized_with_params(const SimpleTensor<int8_t>& mat, 
                                                    const SimpleTensor<half>& vec, 
                                                    unsigned int block_dim_x,
                                                    unsigned int block_dim_y, 
                                                    float scale_f, float zero_point_f) {
  half scale = __float2half(scale_f);
  half zero_point = __float2half(zero_point_f);
  assert(mat.width_ == vec.height_);
  assert(block_dim_y <= SHARED_MEM_MAX_ROWS);
  assert(block_dim_x * block_dim_y <= MAX_THREADS_PER_BLOCK);
  unsigned int num_per_thread = mat.width_ / block_dim_x;
  assert(num_per_thread >= 8);
  SimpleTensor<half> result(vec.height_, 1);
  dim3 grid_dim(1, mat.height_ / block_dim_y);
  dim3 block_dim(block_dim_x, block_dim_y);
  gemv_quantized_int8<<<grid_dim, block_dim>>>(mat.data_, vec.data_, result.data_, 
                                                            mat.width_, scale, zero_point, num_per_thread);
  checkCudaErrors(hipPeekAtLastError());
  return result;
}

SimpleTensor<half> solve_gemv_with_params(const SimpleTensor<half>& mat, 
                                          const SimpleTensor<half>& vec, 
                                          unsigned int block_dim_x,
                                          unsigned int block_dim_y) {
  assert(mat.width_ == vec.height_);
  assert(block_dim_y <= SHARED_MEM_MAX_ROWS);
  assert(block_dim_x * block_dim_y <= MAX_THREADS_PER_BLOCK);
  unsigned int num_per_thread = mat.width_ / block_dim_x;
  assert(num_per_thread >= 8);
  SimpleTensor<half> result(vec.height_, 1);
  dim3 grid_dim(1, mat.height_ / block_dim_y);
  dim3 block_dim(block_dim_x, block_dim_y);
  gemv_fp16<<<grid_dim, block_dim>>>(mat.data_, vec.data_, result.data_,
                                          mat.width_, num_per_thread);
  checkCudaErrors(hipPeekAtLastError());
  return result;
}

///////////////////////////// TEST //////////////////////////////

__global__ void check_correctness(half* mat, half* vec, half* res, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    float result = 0;
    for (int j = 0; j < n; ++j) {
      result += __half2float(mat[idx * n + j]) * __half2float(vec[j]);
    }
    float diff = result - __half2float(res[idx]);
    float delta = 0.125 * n / 512;
    if (diff > delta || diff < -delta) {
      printf("!!![idx=%d] %f != %f, diff=%f\n", idx, __half2float(res[idx]),
             result, diff);
    }
  }
}

__global__ void check_int8_quantized_correctness(int8_t* mat, half* vec, half* res, half scale, half zero_point, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    float result = 0;
    for (int j = 0; j < n; ++j) {
      float dequantized_val = (static_cast<float>(mat[idx * n + j]) - static_cast<float>(zero_point)) * static_cast<float>(scale);
      result += dequantized_val * __half2float(vec[j]);
    }
    float diff = result - __half2float(res[idx]);
    float delta = 0.125 * n / 512;
    if (diff > delta || diff < -delta) {
      printf("!!![idx=%d] %f != %f, diff=%f\n", idx, __half2float(res[idx]),
             result, diff);
    }
  }
}

__global__ void check_int4_quantized_correctness(uint4_2* mat, half* vec, half* res, half scale, half zero_point, int mat_size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < mat_size * 2) {
    float result = 0;
    for (int j = 0; j < mat_size; ++j) {
      uint8_t x = mat[idx * mat_size + j].getX();
      uint8_t y = mat[idx * mat_size + j].getY();
      float dequantized_x = (static_cast<float>(x) - static_cast<float>(zero_point)) * static_cast<float>(scale);
      float dequantized_y = (static_cast<float>(y) - static_cast<float>(zero_point)) * static_cast<float>(scale);
      result += dequantized_x * __half2float(vec[j * 2]);
      result += dequantized_y * __half2float(vec[j * 2 + 1]);
    }
    float diff = result - __half2float(res[idx]);
    float delta = 0.125 * mat_size / 256;
    if (diff > delta || diff < -delta) {
      printf("!!![idx=%d] %f != %f, diff=%f\n", idx, __half2float(res[idx]),
             result, diff);
    }
  }
}

void test_gemv_int4_quantized_with_params(unsigned int size, unsigned int iter, 
                           unsigned int block_dim_x, unsigned int block_dim_y,
                           float scale, float zero_point) {
  hipSetDevice(0);
  // generate data
  const unsigned int mat_width = size / 2;
  SimpleTensor<uint4_2> mat(size, mat_width);
  SimpleTensor<half> vec(size, 1);
  mat.reset();
  vec.reset();

  // compute dot product
  printf("solving...\n");
  SimpleTensor<half> res(size, 1);
  for (int i = 0; i < iter; ++i) {
    res = solve_gemv_int4_quantized_with_params(mat, vec, block_dim_x, block_dim_y, scale, zero_point);
  }

  // check correctness
  printf("checking...\n");
  int threads_per_block = 256;
  int num_blocks = (size + threads_per_block - 1) / threads_per_block;
  check_int4_quantized_correctness<<<num_blocks, threads_per_block>>>(
      mat.device_data(), vec.device_data(), res.device_data(), scale, zero_point, mat_width);
  printf("checked\n");
}

void test_gemv_int8_quantized_with_params(unsigned int size, unsigned int iter, 
                           unsigned int block_dim_x, unsigned int block_dim_y, 
                           float scale, float zero_point) {
  hipSetDevice(0);
  // generate data
  SimpleTensor<int8_t> mat(size, size);
  SimpleTensor<half> vec(size, 1);
  mat.reset();
  vec.reset();

  // compute the dot product
  printf("solving...\n");
  SimpleTensor<half> res(size, 1);

  for (int i = 0; i < iter; ++i) {
    res = solve_gemv_int8_quantized_with_params(mat, vec, block_dim_x, block_dim_y, scale, zero_point);
  }

  // check correctness
  printf("checking...\n");
  int threads_per_block = 256;
  int num_blocks = (size + threads_per_block - 1) / threads_per_block;
  check_int8_quantized_correctness<<<num_blocks, threads_per_block>>>(
      mat.device_data(), vec.device_data(), res.device_data(), scale, zero_point, size);
  printf("checked\n");
}

void test_gemv_with_params(unsigned int size, unsigned int iter,
                           unsigned int block_dim_x, unsigned int block_dim_y) {
  hipSetDevice(0);
  // generate data
  SimpleTensor<half> mat(size, size);
  SimpleTensor<half> vec(size, 1);
  mat.reset();
  vec.reset();

  // compute the dot product
  printf("solving...\n");
  SimpleTensor<half> res(size, 1);

  for (int i = 0; i < iter; ++i) {
    res = solve_gemv_with_params(mat, vec, block_dim_x, block_dim_y);
  }

  // check correctness
  printf("checking...\n");
  int threads_per_block = 256;
  int num_blocks = (size + threads_per_block - 1) / threads_per_block;
  check_correctness<<<num_blocks, threads_per_block>>>(
      mat.device_data(), vec.device_data(), res.device_data(), size);
  printf("checked\n");
}
